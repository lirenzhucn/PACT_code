
#include <hip/hip_runtime.h>
#include <math.h>

#define SIGN(x) ((x) > 0.0 ? 1 : -1)

__global__ void init_image_kernel(float *img) {
  size_t xi = blockIdx.x;
  size_t yi = blockIdx.y;
  size_t zi = threadIdx.x;
  size_t imgIdx = zi + yi*blockDim.x + xi*blockDim.x*gridDim.y;
  img[imgIdx] = 0.0;
}

__global__ void calculate_cos_alpha_and_tempc
(float *cosAlpha, float *tempc, float *xRange, float *yRange,
 float *xReceive, float *yReceive, float lenR) {
  size_t xi = blockIdx.x;
  size_t yi = blockIdx.y;
  size_t ni = threadIdx.x;
  size_t idx = ni + yi*blockDim.x + xi*blockDim.x*gridDim.y;
  float dx = xRange[xi] - xReceive[ni];
  float dy = yRange[yi] - yReceive[ni];
  float r0 = sqrt(xReceive[ni]*xReceive[ni] + yReceive[ni]*yReceive[ni]);
  float rr0 = sqrt(dx*dx + dy*dy);
  cosAlpha[idx] = fabs((-xReceive[ni]*dx-yReceive[ni]*dy)/r0/rr0);
  tempc[idx] = rr0 - lenR/cosAlpha[idx];
}

__global__ void backprojection_kernel_fast
(float *img, float *paDataLine,
 float *cosAlpha_, float *tempc_, float *zRange,
 float zReceive, float lenR, float vm, float delayIdx, float fs,
 unsigned int ni, unsigned int nSteps, unsigned int lineLength) {
  size_t xi = blockIdx.x;
  size_t yi = blockIdx.y;
  size_t zi = threadIdx.x;
  size_t imgIdx = zi + yi*blockDim.x + xi*blockDim.x*gridDim.y;
  size_t precompIdx = ni + yi*nSteps + xi*nSteps*gridDim.y;
  float dz = zRange[zi] - zReceive;
  float cosAlpha = cosAlpha_[precompIdx];
  float tempc = tempc_[precompIdx];
  float rr0 = sqrt(tempc*tempc + dz*dz)*SIGN(tempc) + lenR/cosAlpha;
  if (fabs(dz/tempc) < fabs(10.0/lenR/cosAlpha)) {
    float angleWeightB = tempc/sqrt(tempc*tempc+dz*dz)*cosAlpha/(rr0*rr0);
    size_t idx0 = lround((rr0/vm-delayIdx)*fs);
    if (idx0 < lineLength) {
      img[imgIdx] += paDataLine[idx0] / angleWeightB;
    }
  }
}
