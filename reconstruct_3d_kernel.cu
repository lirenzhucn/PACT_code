
#include <hip/hip_runtime.h>
#include <math.h>

#define SIGN(x) ((x) > 0.0 ? 1 : -1)

__global__ void init_image_kernel(float *img) {
  size_t xi = blockIdx.x;
  size_t yi = blockIdx.y;
  size_t zi = threadIdx.x;
  size_t imgIdx = zi + yi*blockDim.x + xi*blockDim.x*gridDim.y;
  img[imgIdx] = 0.0;
}

__global__ void backprojection_kernel
(float *img, float *paDataLine,
 float *xRange, float *yRange, float *zRange,
 float xReceive, float yReceive, float zReceive,
 float lenR, float vm, float delayIdx, float fs,
 unsigned int lineLength) {
  size_t xi = blockIdx.x;
  size_t yi = blockIdx.y;
  size_t zi = threadIdx.x;
  size_t imgIdx = zi + yi*blockDim.x + xi*blockDim.x*gridDim.y;
  float dx = xRange[xi] - xReceive;
  float dy = yRange[yi] - yReceive;
  float dz = zRange[zi] - zReceive;
  float r0 = sqrt(xReceive*xReceive + yReceive*yReceive);
  float rr0 = sqrt(dx*dx + dy*dy);
  float cosAlpha = fabs((-xReceive*dx-yReceive*dy)/r0/rr0);
  float tempc = rr0 - lenR/cosAlpha;
  rr0 = sqrt(tempc*tempc + dz*dz)*SIGN(tempc) + lenR/cosAlpha;
  if (fabs(dz/tempc) < fabs(10.0/lenR/cosAlpha)) {
    float angleWeightB = tempc/sqrt(tempc*tempc+dz*dz)*cosAlpha/(rr0*rr0);
    size_t idx0 = lround((rr0/vm-delayIdx)*fs);
    if (idx0 < lineLength) {
      img[imgIdx] += paDataLine[idx0] / angleWeightB;
    }
  }
}
